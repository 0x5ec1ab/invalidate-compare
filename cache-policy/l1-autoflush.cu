/******************************************************************************
 * check autoflush behavior of L1
 *****************************************************************************/
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <string.h>

// a chunk consists of a 2MB page
#define CHUNK_SIZE   (2 * 1024 * 1024)

// force each block to reside at one SM
#define BLK_NUM     100
#define SHARED_MEM  (90 * 1024)

// two SMs chosen for this experiment
#define SMID_X      0
#define SMID_Y      12

#define SHORT_DELAY 1000000L
#define MID_DELAY   100000000L
#define LONG_DELAY  10000000000L

// threshold determining if a context switch occurs (RTX 3080's)
#define SW_THR      8000 

__device__ void 
wait_delay(uint64_t delay)
{
  uint64_t start;
  uint64_t diff;
  
  start = clock64();
  do {
    diff = clock64() - start;
  } while (diff < delay);
}

__device__ void 
wait_context_switch()
{
  uint64_t prev;
  uint64_t start;
  uint64_t delta;
  
  prev = 0;
  start = clock64();
  do {
    delta = clock64() - start;
    if (delta - prev > SW_THR)
      break;
    prev = delta;
  } while (1);
}

__global__ void 
check_l1_autoflush(uint64_t addr, uint64_t *vals)
{
  uint64_t temp;
  
  uint32_t smid;
  asm volatile("mov.u32 %0, %%smid;\n\t" : "=r" (smid));
  
  if (smid != SMID_X && smid != SMID_Y)
    return;

  asm volatile(
    ".reg .u64 addr_reg;"
    ".reg .u64 val_reg;"
    "mov.u64 addr_reg, %0;"
    : 
    : "l" (addr)
  );
  
  if (smid == SMID_X) {
    // wait for "Y reads B"
    wait_delay(SHORT_DELAY);
    
    // perform "X writes B"
    asm volatile("st.u64 [addr_reg], 0xdeadbeef;");
    
    // perform "X reads B"
    asm volatile("ld.u64 %0, [addr_reg];" : "=l" (temp));
    
    vals[0] = temp;
  } else {
    // perform "Y reads B"
    asm volatile("ld.u64 val_reg, [addr_reg];");
    
    // wait for "X writes B"
    wait_delay(MID_DELAY);
    
    // wait for context switching back 
    wait_context_switch();
    
    // perform "Y reads B"
    asm volatile("ld.u64 %0, [addr_reg];" : "=l" (temp));
    
    vals[1] = temp;
  }
}

int 
main(int argc, char *argv[])
{
  hipDeviceReset();
  hipFuncSetAttribute(reinterpret_cast<const void*>(check_l1_autoflush), 
      hipFuncAttributeMaxDynamicSharedMemorySize, SHARED_MEM);
    
  uint64_t *data;
  uint64_t *vals;
  uint64_t *host;
  hipMalloc(&data, CHUNK_SIZE);
  hipMalloc(&vals, CHUNK_SIZE);
  hipDeviceSynchronize();
  
  host = (uint64_t *)malloc(CHUNK_SIZE);
  memset(host, 0, CHUNK_SIZE);
  
  hipMemcpy(data, host, CHUNK_SIZE, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  
  uint64_t addr = (uint64_t)data;
  check_l1_autoflush<<<BLK_NUM, 1, SHARED_MEM>>>(addr, vals);
  hipDeviceSynchronize();
  
  hipMemcpy(host, vals, CHUNK_SIZE, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  
  printf("SM Y: value %lx\n", host[1]);
}

