/******************************************************************************
 * check write policy of L1
 *****************************************************************************/
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <string.h>

// a chunk consists of a 2MB page
#define CHUNK_SIZE   (2 * 1024 * 1024)

// force each block to reside at one SM
#define BLK_NUM     100
#define SHARED_MEM  (90 * 1024)

// two SMs chosen for this experiment
#define SMID_X      0
#define SMID_Y      12

#define SHORT_DELAY 1000000L
#define MID_DELAY   100000000L
#define LONG_DELAY  10000000000L

__device__ void 
wait_delay(uint64_t delay)
{
  uint64_t start;
  uint64_t diff;
  
  start = clock64();
  do {
    diff = clock64() - start;
  } while (diff < delay);
}

__global__ void 
check_write(uint64_t addr, uint64_t *vals)
{
  uint64_t temp;
  
  uint32_t smid;
  asm volatile("mov.u32 %0, %%smid;\n\t" : "=r" (smid));
  
  if (smid != SMID_X && smid != SMID_Y)
    return;

  asm volatile(
    ".reg .u64 addr_reg;"
    "mov.u64 addr_reg, %0;"
    : 
    : "l" (addr)
  );
  
  if (smid == SMID_X) {
    // perform "X reads B" 
    asm volatile("ld.u64.ca %0, [addr_reg];" : "=l" (temp));
    
    // perform "X writes B"
    asm volatile("st.u64.wb [addr_reg], 0xdeadbeef;");
    
    // prevent load optimized away
    vals[1] = temp;
  } else {
    // make sure "X writes B" finished
    wait_delay(LONG_DELAY);
    
    // perform "Y reads B"
    asm volatile("ld.u64.ca %0, [addr_reg];" : "=l" (temp));
    
    vals[0] = temp;
  }
}

int 
main(int argc, char *argv[])
{
  hipDeviceReset();
  hipFuncSetAttribute(reinterpret_cast<const void*>(check_write), 
      hipFuncAttributeMaxDynamicSharedMemorySize, SHARED_MEM);
    
  uint64_t *data;
  uint64_t *vals;
  uint64_t *host;
  hipMalloc(&data, CHUNK_SIZE);
  hipMalloc(&vals, CHUNK_SIZE);
  hipDeviceSynchronize();
  
  host = (uint64_t *)malloc(CHUNK_SIZE);
  memset(host, 0, CHUNK_SIZE);
  
  hipMemcpy(data, host, CHUNK_SIZE, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  
  uint64_t addr = (uint64_t)data;
  check_write<<<BLK_NUM, 1, SHARED_MEM>>>(addr, vals);
  hipDeviceSynchronize();

  hipMemcpy(host, vals, CHUNK_SIZE, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  printf("SM Y: value %lx\n", host[0]);
}

