/******************************************************************************
 * check write-allocate policy of L2
 *****************************************************************************/
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <string.h>

// a chunk consists of a 2MB page
#define CHUNK_SIZE   (2 * 1024 * 1024)

// force each block to reside at one SM
#define BLK_NUM     100
#define SHARED_MEM  (90 * 1024)

// two SMs chosen for this experiment
#define SMID_X      0
#define SMID_Y      12

#define SHORT_DELAY 1000000L
#define MID_DELAY   100000000L
#define LONG_DELAY  10000000000L

__device__ void 
wait_delay(uint64_t delay)
{
  uint64_t start;
  uint64_t diff;
  
  start = clock64();
  do {
    diff = clock64() - start;
  } while (diff < delay);
}

__global__ void 
check_l2_allocate(uint64_t addr, uint64_t *vals)
{
  uint64_t temp;
  
  uint32_t smid;
  asm volatile("mov.u32 %0, %%smid;\n\t" : "=r" (smid));
  
  if (smid != SMID_X && smid != SMID_Y)
    return;

  asm volatile(
    ".reg .u64 addr_reg;"
    "mov.u64 addr_reg, %0;"
    : 
    : "l" (addr)
  );
  
  if (smid == SMID_X) {
    // perform "X reads B"
    asm volatile("ld.u64 %0, [addr_reg];" : "=l" (temp));
    
    vals[0] = temp;
    
    // make sure "Y discards B" finished
    wait_delay(LONG_DELAY);
    
    // perform "X writes B"
    asm volatile("st.u64 [addr_reg], 0xdeadbeef;");
    
    // perform "X discards B"
    asm volatile("discard.global.L2 [addr_reg], 128;");
    
    // perform "X reads B"
    asm volatile("ld.u64 %0, [addr_reg];" : "=l" (temp));
    
    vals[1] = temp;
  } else {
    // perform "Y reads B"
    asm volatile("ld.u64 %0, [addr_reg];" : "=l" (temp));
    
    // make sure "X reads B" finished
    wait_delay(MID_DELAY);
    
    // perform "Y discards B"
    asm volatile("discard.global.L2 [addr_reg], 128;");
    
    vals[2] = temp;
  }
}

int 
main(int argc, char *argv[])
{
  hipDeviceReset();
  hipFuncSetAttribute(reinterpret_cast<const void*>(check_l2_allocate), 
      hipFuncAttributeMaxDynamicSharedMemorySize, SHARED_MEM);
    
  uint64_t *data;
  uint64_t *vals;
  uint64_t *host;
  hipMalloc(&data, CHUNK_SIZE);
  hipMalloc(&vals, CHUNK_SIZE);
  hipDeviceSynchronize();
  
  host = (uint64_t *)malloc(CHUNK_SIZE);
  memset(host, 0, CHUNK_SIZE);
  
  hipMemcpy(data, host, CHUNK_SIZE, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  
  uint64_t addr = (uint64_t)data;
  check_l2_allocate<<<BLK_NUM, 1, SHARED_MEM>>>(addr, vals);
  hipDeviceSynchronize();

  hipMemcpy(host, vals, CHUNK_SIZE, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  printf("SM X: 1st value %lx\n", host[0]);
  printf("SM X: 2nd value %lx\n", host[1]);
  
  free(host);
  hipFree(vals);
  hipFree(data);
}

